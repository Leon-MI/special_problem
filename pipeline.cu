#include "hip/hip_runtime.h"
#include <iostream>

#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/core/utility.hpp>
#include <hip/hip_runtime.h>
#include <fstream>
#include <cmath>

using namespace cv;
using namespace cv::cuda;

#define ROWS 2048
#define COLS 2448
#define ROWS2 1024
#define COLS2 1224

#define ALLOC_TYPE AllocType::unified

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__global__ void my_split(const cuda::PtrStepSzb dev_img, cuda::PtrStepSzb bggr0, cuda::PtrStepSzb bggr45, cuda::PtrStepSzb bggr90, cuda::PtrStepSzb bggr135) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= ROWS || j >= COLS) {
        return;
    }
    if (i%2 == 0) {
        if (j%2 == 0)
            bggr90(i/2, j/2) = dev_img(i, j);
        else
            bggr45(i/2, (j-1)/2) = dev_img(i, j);
    } else {
        if (j%2 == 0)
            bggr135((i-1)/2, j/2) = dev_img(i, j);
        else
            bggr0((i-1)/2, (j-1)/2) = dev_img(i, j);
    }
}


__global__ void compute_stokes(
    const cuda::PtrStepSzb mono0,
    const cuda::PtrStepSzb mono45,
    const cuda::PtrStepSzb mono90,
    const cuda::PtrStepSzb mono135,
    cuda::PtrStep<int3>  output
    ) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    const int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i > (ROWS2-1) || j > (COLS2-1))
        return;

    const unsigned char m0 = mono0(i, j);
    const unsigned char m90 = mono90(i, j);
    
    output(i, j).x = (int)(m0 + m90);
    output(i, j).y = (int)(m0 - m90);
    output(i, j).z = (int)(mono45(i, j) - mono135(i, j));
}

__global__ void compute_dolp(const cuda::PtrStep<int3> stokes, cuda::PtrStepSz<float> output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i > (ROWS2-1) || j > (COLS2-1))
        return;
    
    const int3 s = stokes(i, j);
    if (s.x == 0)
        output(i, j) = 0;
    else
        output(i, j) = sqrtf((s.y*s.y) + (s.z*s.z)) / s.x;
}


__global__ void compute_aolp(const cuda::PtrStep<int3> stokes, cuda::PtrStepSz<float> output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i > (ROWS2-1) || j > (COLS2-1))
        return;

    const int3 s = stokes(i, j);
    if (s.z == (int)0) {
        output(i, j) = 0.0;
    } else {
        const float sy = (float)s.y;
        const float sz = (float)s.z;
        const float angle = ((float)1/2) * atan2f(sy, sz);
        output(i, j) = angle + (float)CV_PI/2;
    }
}

__global__ void false_coloring(const cuda::PtrStepSz<float> aolp, cuda::PtrStepSz<float> dolp, cuda::PtrStep<uchar3> output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i > (ROWS2-1) || j > (COLS2-1))
        return;

    const float a = aolp(i, j);
    const float d = dolp(i, j) * 255;

    output(i, j).x = (unsigned char) (179 * fmodf(a, CV_PI) / CV_PI);
    output(i, j).y = (unsigned char) 255;
    output(i, j).z = (unsigned char) min(max((double)0, d), (double)255);
}


inline size_t imageFormatSize(size_t width, size_t height, int format)
{
    size_t s = sizeof(uchar3);
    if (format == CV_8UC1)
        s = sizeof(uchar1);
    else if (format == CV_32FC3)
        s = sizeof(float3);
    else if (format == CV_32FC1)
        s = sizeof(float);
    else if (format == CV_64FC1)
        s = sizeof(double);
    else if (format == CV_32SC3)
        s = sizeof(int3);

	return width * height * s;
}

enum AllocType {
    splitted,
    shared,
    mapped, 
    unified
};

class MMat {
    private:
        AllocType alloc_type;
        cuda::HostMem hostMem;
    public:
        Mat cpuMat;
        GpuMat gpuMat;

        MMat(int rows, int cols, int img_type, AllocType alloc_type_) {
            alloc_type = alloc_type_;
            if (alloc_type == AllocType::splitted) {
                gpuMat = GpuMat(rows, cols, img_type);
            } else if (alloc_type == AllocType::shared) {
                hostMem = cuda::HostMem(rows, cols, img_type, cuda::HostMem::SHARED );
                CHECK_LAST_CUDA_ERROR();
                cpuMat = hostMem.createMatHeader();
                gpuMat = hostMem.createGpuMatHeader();
            } else if (alloc_type == AllocType::mapped) {
                void *cpu_ptr, *gpu_ptr;
                size_t size = imageFormatSize(cols, rows, img_type);
                hipHostAlloc(&cpu_ptr, size, hipHostMallocMapped);
                CHECK_LAST_CUDA_ERROR();
                hipHostGetDevicePointer(&gpu_ptr, cpu_ptr, 0);
                CHECK_LAST_CUDA_ERROR();

                gpuMat = GpuMat(rows, cols, img_type, gpu_ptr);
                cpuMat = Mat(rows, cols, img_type, cpu_ptr);
            } else if (alloc_type == AllocType::unified) {
                void* unified_ptr;
                size_t size = imageFormatSize(cols, rows, img_type);
                hipMallocManaged(&unified_ptr, size);
                CHECK_LAST_CUDA_ERROR();

                cpuMat = Mat(rows, cols, img_type, unified_ptr);
                gpuMat = GpuMat(rows, cols, img_type, unified_ptr);
            }
        }

        ~MMat() {
            switch (alloc_type) {
                case AllocType::splitted:
                    // TODO Check if cpuMat is initialized
                    cpuMat.release();
                    gpuMat.release();
                    break;
                case AllocType::shared:
                case AllocType::mapped:
                case AllocType::unified:
                    cpuMat.release();
                    break;
                default:
                    break;
            }
        }

        void download() {
            if (alloc_type == AllocType::splitted)
                gpuMat.download(cpuMat);
        }

        void download(cuda::Stream & s) {
            if (alloc_type == AllocType::splitted)
                gpuMat.download(cpuMat, s);
        }

        void save_img(const char * name) {
            download();
            std::string out_name = name;
            out_name = "images/" + out_name + ".png";
            std::vector<int> compression_params;
            compression_params.push_back(IMWRITE_PNG_COMPRESSION);
            compression_params.push_back(9);
            imwrite(out_name, cpuMat, compression_params);
        }

        MMat(const MMat&) = delete;
        // "copy assignment operator"
        MMat& operator= (const MMat&) = delete;  //  MMat p6; p6 = p1;
        // "move constructor"
        MMat(MMat&&) = delete;                   //  MMat p7{ std::move(p2) };
        // "move assignment operator"
        MMat& operator= (MMat&&) = delete;  
};



void benchmark_indiv(const GpuMat & dev_img_raw) {
    const int n = 1;
    float t_upload = 0.0f;
    float t_split = 0.0f;
    float t_debayer_mono = 0.0f;
    float t_stokes = 0.0f;
    float t_aolp_dolp = 0.0f;
    float t_hsv_rgb = 0.0f;
    float elapsed_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    MMat bggr0 (ROWS2, COLS2, CV_8UC1, ALLOC_TYPE );
    MMat bggr45 (ROWS2, COLS2, CV_8UC1, ALLOC_TYPE );
    MMat bggr90 (ROWS2, COLS2, CV_8UC1, ALLOC_TYPE );
    MMat bggr135 (ROWS2, COLS2, CV_8UC1, ALLOC_TYPE );

    MMat rgb0 (ROWS2, COLS2, CV_8UC3, ALLOC_TYPE);
    MMat rgb45 (ROWS2, COLS2, CV_8UC3, ALLOC_TYPE);
    MMat rgb90 (ROWS2, COLS2, CV_8UC3, ALLOC_TYPE);
    MMat rgb135 (ROWS2, COLS2, CV_8UC3, ALLOC_TYPE);
    MMat mono0(ROWS2, COLS2, CV_8UC1, ALLOC_TYPE);
    MMat mono45(ROWS2, COLS2, CV_8UC1, ALLOC_TYPE);
    MMat mono90(ROWS2, COLS2, CV_8UC1, ALLOC_TYPE);
    MMat mono135(ROWS2, COLS2, CV_8UC1, ALLOC_TYPE);

    MMat m_stokes(ROWS2, COLS2, CV_32SC3, ALLOC_TYPE);

    MMat dolp(ROWS2, COLS2, CV_32FC1, ALLOC_TYPE);
    MMat aolp(ROWS2, COLS2, CV_32FC1, ALLOC_TYPE);

    MMat hsv(ROWS2, COLS2, CV_8UC3, ALLOC_TYPE);
    MMat colored (ROWS2, COLS2, CV_8UC3, ALLOC_TYPE);

    CHECK_LAST_CUDA_ERROR();

    for (int i = 0; i < n; i++) {
        // Split
        dim3 blocks(64, 77);
        dim3 threads(32, 32);
        hipEventRecord(start, 0);
        my_split<<<blocks, threads>>>(dev_img_raw, bggr0.gpuMat, bggr45.gpuMat, bggr90.gpuMat, bggr135.gpuMat);
        CHECK_LAST_CUDA_ERROR();

        bggr0.download(); bggr45.download(); bggr90.download(); bggr135.download();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        t_split += elapsed_time;

        // Debayer + RGB To mono
        blocks.x=32;
        blocks.y=39;

        cuda::Stream s0, s45, s90, s135;
        hipEventRecord(start, 0);
        cuda::cvtColor(bggr0.gpuMat, rgb0.gpuMat, COLOR_BayerBG2BGR, 0, s0);
        cuda::cvtColor(bggr45.gpuMat, rgb45.gpuMat, COLOR_BayerBG2BGR, 0, s45);
        cuda::cvtColor(bggr90.gpuMat, rgb90.gpuMat, COLOR_BayerBG2BGR, 0, s90);
        cuda::cvtColor(bggr135.gpuMat, rgb135.gpuMat, COLOR_BayerBG2BGR, 0, s135);
        rgb0.download(s0); rgb45.download(s45); rgb90.download(s90); rgb135.download(s135);

        cuda::cvtColor(rgb0.gpuMat, mono0.gpuMat, COLOR_BGR2GRAY, 0, s0);
        cuda::cvtColor(rgb45.gpuMat, mono45.gpuMat, COLOR_BGR2GRAY, 0, s45);
        cuda::cvtColor(rgb90.gpuMat, mono90.gpuMat, COLOR_BGR2GRAY, 0, s90);
        cuda::cvtColor(rgb135.gpuMat, mono135.gpuMat, COLOR_BGR2GRAY, 0, s135);
        mono0.download(s0); mono45.download(s45); mono90.download(s90); mono135.download(s135);

        s0.waitForCompletion();
        s45.waitForCompletion();
        s90.waitForCompletion();
        s135.waitForCompletion();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        t_debayer_mono += elapsed_time;

        // Stokes
        hipEventRecord(start, 0);
        compute_stokes<<<blocks, threads>>>(mono0.gpuMat, mono45.gpuMat, mono90.gpuMat, mono135.gpuMat, m_stokes.gpuMat);
        m_stokes.download();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        t_stokes += elapsed_time;

        // Dolp + Aolp
        hipStream_t sdolp, saolp; 
        hipStreamCreate(&sdolp);
        hipStreamCreate(&saolp);
    
        hipEventRecord(start, 0);
        compute_dolp<<<blocks, threads, 0, sdolp>>>(m_stokes.gpuMat, dolp.gpuMat);
        compute_aolp<<<blocks, threads, 0, saolp>>>(m_stokes.gpuMat, aolp.gpuMat);

        hipStreamDestroy(sdolp);
        hipStreamDestroy(saolp);
        dolp.download();
        aolp.download();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        t_aolp_dolp += elapsed_time;

        // False coloring
        hipEventRecord(start, 0);
        false_coloring<<<blocks, threads>>>(aolp.gpuMat, dolp.gpuMat, hsv.gpuMat);

        cuda::cvtColor(hsv.gpuMat, colored.gpuMat, COLOR_HSV2RGB);
        colored.download();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        t_hsv_rgb += elapsed_time;
    }

    colored.save_img("colored");

    hipEventDestroy(start);
    hipEventDestroy(stop);
    std::cout << "Turtlebot, mean over " << n << " runs" << std::endl;
    std::cout << "Host/Device Memory, streams on debayer/mono & aolp/dolp" << std::endl; 
    std::cout << "Stokes: CV_32FC3, AOLP/DOLP: CV_32FC1" << std::endl; 
    std::cout << "Commit :" << std::endl << std::endl;
    std::cout << "Upload: " << t_upload / n << "ms" << std::endl;
    std::cout << "Split: " << t_split/n << "ms" << std::endl;
    std::cout << "Debayer + mono: " << t_debayer_mono / n << "ms" << std::endl;
    std::cout << "Stokes: " << t_stokes / n << "ms" << std::endl;
    std::cout << "Aolp + Dolp: " << t_aolp_dolp / n << "ms" << std::endl;
    std::cout << "False coloring: " << t_hsv_rgb / n << "ms" << std::endl;
    std::cout << "Total: " << (t_upload + t_split + t_debayer_mono + t_stokes + t_aolp_dolp + t_hsv_rgb) / n << "ms" << std::endl; 
}

int main()
{
    cuda::setDevice(0);

    // hipSetDeviceFlags(hipDeviceMapHost);
    // Read and upload img to gpu
    Mat img_raw = imread("images/frame00000_raw.png", IMREAD_GRAYSCALE);
    std::cout << img_raw.type() << std::endl;
    GpuMat dev_img_raw;
    dev_img_raw.upload(img_raw);

    benchmark_indiv(dev_img_raw);

    dev_img_raw.release();
    img_raw.release();
    return 0;
}
